#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

//---------------------------------------------------------------------------------
static const int N = 1000001; //Number of rows in input matrix
static const int M = 100; //Number of columns in input matrix

using namespace std;
//---------------------------------------------------------------------------------
/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
//---------------------------------------------------------------------------------
__global__ void matrixTranspose(unsigned int* A_d, unsigned int *T_d, int rowCount, int colCount) {

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** Populate vecADD kernel function ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < rowCount && col < colCount){
		T_d[col*rowCount+row] = A_d[row*colCount+col];
	}

}


//---------------------------------------------------------------------------------
int main(void) {
	unsigned int **A ;
	unsigned int **T ;
	unsigned int *A_h;
	unsigned int *A_d;
	unsigned int *T_h;
	unsigned int *T_d;

	//Set Device
	CUDA_CHECK_RETURN(hipSetDevice(0));

	//See random number generator
	srand(time(NULL));

	//Clear command prompt
	cout << "\033[2J\033[1;1H";

	cout << "Allocating arrays on host ... ";
	A_h = new unsigned int[N*M];
	T_h = new unsigned int[N*M];

	A = new unsigned int* [N];
	for (int i = 0; i < N; ++i) {
		A[i] = new unsigned int[M];
	}

	T = new unsigned int* [M];
	for (int i = 0; i < M; ++i) {
		T[i] = new unsigned int[N];
	}

	cout << "done.\nPopluating input matrix on host ... ";
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			A[i][j] = rand();
		}
	}

	cout << "done.\nConverting 2-dimensional input matrix to 1-dimensional array on host ... ";

    //@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** Add code for converting 2-dimensional input matrix to 1-dimensional array here  ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			A_h[i*M+j] = A[i][j];
		}
	}

	cout << "done.\nAllocating arrays on device ... ";
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &A_d, sizeof(unsigned int) * N*M));
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &T_d, sizeof(unsigned int) * N*M));

	cout << "done.\nCopying arrays from host to device ... ";
	CUDA_CHECK_RETURN(
			hipMemcpy(A_d, A_h, sizeof(int) * N*M,
					hipMemcpyHostToDevice));

	cout << "done.\nLaunching kernel ... ";

    //@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** define kernel launch parameters ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	dim3 dimBlock(32,32);
	dim3 dimGrid(ceil((double)M/32), ceil((double)N/32));

	//Time kernel launch
	//Time kernel launch
	hipEvent_t start, stop;
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
	float elapsedTime;

	CUDA_CHECK_RETURN(hipEventRecord(start, 0));



    //@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** Add kernel call here ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	matrixTranspose<<< dimGrid, dimBlock >>>(A_d, T_d, N, M);

	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));

	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError()); //Check if an error occurred in device code
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	cout << "done.\nElapsed kernel time: " << elapsedTime << " ms\n";

	cout << "Copying results back to host .... ";
	CUDA_CHECK_RETURN(
			hipMemcpy(T_h, T_d, sizeof(int) * N*M,
					hipMemcpyDeviceToHost));

	cout << "done.\nConverting 1-dimensional output array to 2-dimensional matrix on host ... ";

    //@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	//**** Add code for converting 1-dimensional output array to 2-dimensional matrix here  ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	for (int i = 0; i < M; ++i) {
		for (int j = 0; j < N; ++j) {
			T[i][j] = T_h[i*N+j];
		}
	}

	cout << "done.\nVerifying results on host ... ";

	//Add code to time host calculations

	clock_t st, ed;

	st = clock();

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
    // **** Check that results from kernel are correct ****
    // **** Complete validation code below             ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@


	bool valid = true;

	for (int i = 0; i < M; i++){
		for(int j = 0; j < N; j++){
			if (T[i][j] != A[j][i])
			{
				cout << "done.\n***GPU results are incorrect***";
				valid = false;
				break;
			}
		}
		if(!valid){
			break;
		}
	}

	cout << "done\n";

	if (valid) {
		cout << "GPU results are valid.\n";
	}


    ed = clock() - st;
	cout << "Elapsed time on host: " << ((float) ed) / CLOCKS_PER_SEC * 1000
			<< " ms" << endl;

	cout << "Freeing memory on device ... ";
	CUDA_CHECK_RETURN(hipFree((void* ) A_d));
	CUDA_CHECK_RETURN(hipFree((void* ) T_d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	cout << "done.\nFreeing memory on host ... ";
	delete[] A_h;
	delete[] T_h;

	for (int i = 0; i < N; ++i) {
		delete[] A[i];
	}
	delete[] A;

	for (int i = 0; i < M; ++i) {
		delete[] T[i];
	}
	delete[] T;

	cout << "done.\nExiting program.\n";
	return 0;
}

